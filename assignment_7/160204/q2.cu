
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);

#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

#define USAGE_EXIT(s) do{ \
                             printf("Usage: %s <# of elements> <# of threads> \n %s\n", argv[0], s); \
                            exit(-1);\
                    }while(0);


__global__ void xorsum(int num_elements,int *a,double l)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= num_elements)
       return;
  int k=(int)l+1;
  int p=num_elements/2;
  int num=num_elements;
  while(p)
  {
     if(i<p)
     {
       a[i]=a[i]^a[num-i-1];
     }
     num=(num+1)/2;
     p=num/2;
  }

}
int main(int argc, char const *argv[])
{
  struct timeval start, end, t_start, t_end;
  int num_elements;
  int SEED,ctr,blocks;
  int *ptr;
  int *gpu_mem,*a;
  if(argc==3)
  {
    num_elements=atoi(argv[1]);
    SEED=atoi(argv[2]);
  }
  else
  {
      printf("Wrong command line arguments\n" );
      exit(-1);
  }

  a = (int *)malloc(num_elements * sizeof(int));
  if(!a){
          USAGE_EXIT("invalid num elements, not enough memory");
  }

  srand(SEED);
  for(ctr=0; ctr<num_elements; ++ctr)
  {
    a[ctr] = random();
  }
  for (size_t i = 0; i < num_elements; i++) 
  {
    printf("%d\n",a[i] );
    if (i==num_elements-1) 
    {
      printf("\n" );    
    }
  }
  gettimeofday(&t_start,NULL);
  hipMalloc(&gpu_mem, num_elements*sizeof(int));
  CUDA_ERROR_EXIT("hipMalloc");

  hipMemcpy(gpu_mem,a, num_elements*sizeof(int) , hipMemcpyHostToDevice);
  CUDA_ERROR_EXIT("hipMemcpy");

  gettimeofday(&start, NULL);

  blocks = num_elements /1024;

  if(num_elements % 1024)
  {
    ++blocks;
  }
  double l=log(num_elements)/log(2);
  xorsum<<<blocks, 1024>>>(num_elements,gpu_mem,l);
  CUDA_ERROR_EXIT("kernel invocation");
  gettimeofday(&end, NULL);

    /* Copy back result*/

  hipMemcpy(a, gpu_mem,num_elements*sizeof(int),hipMemcpyDeviceToHost);
  CUDA_ERROR_EXIT("memcpy");
  gettimeofday(&t_end, NULL);

  printf("Xor based checksum is %d\n",a[0]);
  printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
  hipFree(gpu_mem);
  return 0;
}


