
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    double num1;
                    double num2;
                    double result;
};



__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    return;
}
__global__ void calculate(char *mem, int num,int rows,int blocks)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      //int i = threadIdx.x;
      int j =  threadIdx.y;
      if(i >= num)
           return;
      struct num_array *a = (struct num_array *)(mem +3*(rows*blocks*j+i)*sizeof(double));
      function(a);
}
int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;
    unsigned long num = NUM,rows,cols;   /*Default value of num from MACRO*/
    int blocks;

    if(argc == 4){
         num  = atoi(argv[1]);
         rows = atoi(argv[2]);
         cols = atoi(argv[3]);
         if(num <= 0)
               num = NUM;
    }

    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr;
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }


    gettimeofday(&t_start, NULL);

    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");


    dim3 threadsperblock(rows,cols);


    gettimeofday(&start, NULL);

    blocks = num /1024;
    if(num % 1024)
           ++blocks;

    calculate<<<blocks, threadsperblock>>>(gpu_mem, num,rows,blocks);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);

    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;

    /*Print the last element for sanity check*/
    
	pa = (struct num_array *) (sptr + (num-1)*3*sizeof(double));
	printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);

    free(ptr);
}



